#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <algorithm>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "utils.h"

__global__ void AddVectors(float *a, float *b, float *c, const size_t elementNum)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx<elementNum)
		c[idx] = a[idx] + b[idx];
}

void AddVectorsHost(float *a, float *b, float *c, const size_t elementNum)
{
	for(size_t i=0; i<elementNum; i++)
	{
		c[i] = a[i] + b[i];
	}
}
int main()
{
	constexpr size_t STREAMS_NUM = 2;
	constexpr size_t VECTOR_SIZE = 40960;
	constexpr size_t FULL_SIZE = VECTOR_SIZE * STREAMS_NUM;
	constexpr size_t THREADS_PER_BLOCK = 256;
	float *a_host, *b_host, *c_host, *res_host;
	float *a_dev[STREAMS_NUM], *b_dev[STREAMS_NUM], *c_dev[STREAMS_NUM];

	hipEvent_t start, stop;
	hipStream_t streams[STREAMS_NUM];

	// Alloc host data
	//a_host = (float*)malloc(sizeof(float) * VECTOR_SIZE * STREAMS_NUM);
	checkCudaErrors(hipHostMalloc(&a_host, sizeof(float) * VECTOR_SIZE * STREAMS_NUM));
	//b_host = (float *)malloc(sizeof(float) * VECTOR_SIZE * STREAMS_NUM);
	checkCudaErrors(hipHostMalloc(&b_host, sizeof(float) * VECTOR_SIZE * STREAMS_NUM));
	//c_host = (float *)malloc(sizeof(float) * VECTOR_SIZE * STREAMS_NUM);
	checkCudaErrors(hipHostMalloc(&c_host, sizeof(float) * VECTOR_SIZE * STREAMS_NUM));
	//res_host = (float *)malloc(sizeof(float) * VECTOR_SIZE * STREAMS_NUM);
	checkCudaErrors(hipHostMalloc(&res_host, sizeof(float) * VECTOR_SIZE * STREAMS_NUM));

	// Init host data
	InitData(a_host, b_host, c_host, VECTOR_SIZE * STREAMS_NUM);

	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	for (int i = 0; i < STREAMS_NUM; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));
	
	checkCudaErrors(hipEventRecord(start, 0));

	for (int i = 0; i < STREAMS_NUM; i++)
	{
		// Alloc device data
		checkCudaErrors(hipMallocAsync(&a_dev[i], sizeof(float) * VECTOR_SIZE, streams[i]));
		checkCudaErrors(hipMallocAsync(&b_dev[i], sizeof(float) * VECTOR_SIZE, streams[i]));
		checkCudaErrors(hipMallocAsync(&c_dev[i], sizeof(float) * VECTOR_SIZE, streams[i]));

		//Copy host data to dev
		checkCudaErrors(hipMemcpyAsync(a_dev[i], a_host + i, sizeof(float) * VECTOR_SIZE, hipMemcpyHostToDevice, streams[i]));
		checkCudaErrors(hipMemcpyAsync(b_dev[i], b_host + i, sizeof(float) * VECTOR_SIZE, hipMemcpyHostToDevice, streams[i]));
		checkCudaErrors(hipMemsetAsync(c_dev[i], 0, sizeof(float) * VECTOR_SIZE, streams[i]));

		//Compute device
		dim3 blocks(1,1,1);
		dim3 threads(1,1,1);

		threads.x = THREADS_PER_BLOCK;
		blocks.x = (VECTOR_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		AddVectors<<<blocks, threads, 0, streams[i]>>>(a_dev[i], b_dev[i], c_dev[i], VECTOR_SIZE);

		hipError_t errorNum = hipGetLastError();
		if(errorNum != hipSuccess)
			printf("Cuda error running kernel.\n%s\n", hipGetErrorName(errorNum));
		
		//Copy data back to host
		checkCudaErrors(hipMemcpyAsync(res_host+i, c_dev[i], sizeof(float) * VECTOR_SIZE, hipMemcpyDeviceToHost, streams[i]));

		//Free device data
		checkCudaErrors(hipFreeAsync(a_dev[i], streams[i]));
		checkCudaErrors(hipFreeAsync(b_dev[i], streams[i]));
		checkCudaErrors(hipFreeAsync(c_dev[i], streams[i]));
	}
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float elapsedTime = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Cuda execution time: %f ms.\n", elapsedTime);

	for (int i = 0; i < STREAMS_NUM; i++)
		checkCudaErrors(hipStreamDestroy(streams[i]));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	// Compute host
	AddVectorsHost(a_host, b_host, c_host, VECTOR_SIZE);

	//Check results
	float maxDiff = CheckResults(c_host, res_host, VECTOR_SIZE);

	printf("Max difference between DEVICE and HOST: %f\n", maxDiff);

	//Free host data
	//free(a_host);
	checkCudaErrors(hipHostFree(a_host));
	//free(b_host);
	checkCudaErrors(hipHostFree(b_host));
	//free(c_host);
	checkCudaErrors(hipHostFree(c_host));
	//free(res_host);
	checkCudaErrors(hipHostFree(res_host));
	return 0;
}