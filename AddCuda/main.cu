#include <cstdio>

#include <hip/hip_runtime.h>

//kernel definition
__global__ void Add(float a, float b, float *c)
{
	*c = a + b;
}

int main()
{
	float *c_host, *c_dev;
	
	float a = 1.2;
	float b = 3.2;

	//allocate data
	c_host = (float*)malloc(sizeof(float));
	hipMalloc(&c_dev, sizeof(float));
	
	//kernel lunch
	Add<<<1,1>>>(a, b, c_dev);
	
	hipMemcpy(c_host, c_dev, sizeof(float), hipMemcpyDeviceToHost);
	
	printf("%f + %f = %f\n", a, b, *c_host);

	//free data
	hipFree(c_dev);
	free(c_host);
	return 0;
}