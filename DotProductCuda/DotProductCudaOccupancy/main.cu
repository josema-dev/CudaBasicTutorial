#include "hip/hip_runtime.h"
#include <cstdio>
#include <cmath>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "utils.h"

__global__ void Dot(float *a, float *b, float *c, size_t elementsNum)
{
	//shared memory
	extern __shared__ float blockData[];

	//local memory
	size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	size_t cacheIdx = threadIdx.x;
	float tmp = 0.0f;

	if(idx < elementsNum)
	{
		tmp += a[idx] * b[idx];
	}
	blockData[cacheIdx] = tmp;
	
	//sync all threads in block
	__syncthreads();

	//threads per block must be a power of 2
	int i = blockDim.x / 2;
	while(i != 0)
	{
		if(cacheIdx < i)
		{
			blockData[cacheIdx] += blockData[cacheIdx + i];
		}
		__syncthreads();
		i /= 2;
	}

	if(cacheIdx == 0)
	{
		c[blockIdx.x] = blockData[0];
	}
}

float DotHost(float *a, float *b, size_t elementsNum)
{
	float res = 0.0f;
	for (size_t i = 0; i < elementsNum; i++)
	{
		res += (a[i] * b[i]);
	}

	return res;
}

int main()
{
	constexpr size_t THREADS_PER_BLOCK = 32;
	constexpr size_t VECTOR_SIZE = 256;

	float *a_host, *b_host, *res_host;
	float *a_dev, *b_dev, *c_dev;

	a_host = (float *)malloc(sizeof(float) * VECTOR_SIZE);
	b_host = (float *)malloc(sizeof(float) * VECTOR_SIZE);
	res_host = (float *)malloc(sizeof(float) * VECTOR_SIZE);
	
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("Multiprocessoor count: %d\n", deviceProp.multiProcessorCount);

	checkCudaErrors(hipMalloc(&a_dev, sizeof(float) * VECTOR_SIZE));
	checkCudaErrors(hipMalloc(&b_dev, sizeof(float) * VECTOR_SIZE));

	InitData(a_host, b_host, nullptr, VECTOR_SIZE);

	checkCudaErrors(hipMemcpy(a_dev, a_host, sizeof(float) * VECTOR_SIZE, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(b_dev, b_host, sizeof(float) * VECTOR_SIZE, hipMemcpyHostToDevice));

	dim3 threads, blocks;
	threads.x = THREADS_PER_BLOCK;
	blocks.x = (VECTOR_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	
	checkCudaErrors(hipMalloc(&c_dev, sizeof(float) * blocks.x));
	size_t dynamicSmemSize = 0;
	checkCudaErrors(cudaOccupancyAvailableDynamicSMemPerBlock(&dynamicSmemSize, &Dot, blocks.x, threads.x));
	printf("Shared memeory max size per multiprocessor: %zd\n", dynamicSmemSize);

	int numBlocks = 0;
	checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, &Dot, threads.x, THREADS_PER_BLOCK * sizeof(float)));
	printf("Max blocks size per multiprocessor: %d\n", numBlocks);

	int minGridSize = 0;
	int blockSize = 0;
	checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &Dot, THREADS_PER_BLOCK * sizeof(float)));
	printf("Min grid size: %d, block size: %d\n", minGridSize, blockSize);

	printf("Blocks number: %d\nSet blocks: %d, threads: %d\n", numBlocks * deviceProp.multiProcessorCount, blocks.x, threads.x);
	Dot<<<blocks, threads, THREADS_PER_BLOCK * sizeof(float)>>>(a_dev, b_dev, c_dev, VECTOR_SIZE);

	checkCudaErrors(hipMemcpy(res_host, c_dev, sizeof(float) * blocks.x, hipMemcpyDeviceToHost));

	float dot_host = DotHost(a_host, b_host, VECTOR_SIZE);

	float c = 0;
 	for (size_t i=0; i<blocks.x; i++) {
 		c += res_host[i];
 	}

	printf("CPU res: %f, GPU res: %f, diff: %f\n", dot_host, c, std::abs(c - dot_host));
	return 0;
}