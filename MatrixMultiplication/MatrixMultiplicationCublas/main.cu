#include <cstdio>
#include <vector>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

#include "utils.h"

// rowsA: number of rows of matrix A nad C
// colsA: number of columns matrix B and C
// k1: number of columns of A and rows B
void MatrixMulHost(float *A, float *B, float *C, const size_t rowsA, const size_t colsA, const size_t colsB)
{
	for (size_t i = 0; i < rowsA; i++)
	{
		for (size_t j = 0; j < colsB; j++)
		{
			float sum = 0;
			size_t c_idx = i * colsB + j;
			for (size_t k = 0; k < colsA; k++)
			{
				size_t b_idx = k * colsB + j;
				size_t a_idx = i * colsA + k;
				 sum += B[b_idx] * A[a_idx];
				//printf("a[%d] = %f, b[%d] = %f, c[%d] = %f\n", a_idx, A[a_idx], b_idx, B[b_idx], c_idx, C[c_idx]);
			}
			C[c_idx] = sum;
		}
	}
}

int main()
{
	constexpr size_t MATRIX_ROW_A = 640;
	constexpr size_t MATRIX_COL_A = 480;
	constexpr size_t MATRIX_ROW_B = MATRIX_COL_A;
	constexpr size_t MATRIX_COL_B = 320;
	constexpr size_t MATRIX_ROW_C = MATRIX_ROW_A;
	constexpr size_t MATRIX_COL_C = MATRIX_COL_B;

	float *aMatrixHost, *bMatrixHost, *cMatrixHost, *resMatHost;

	aMatrixHost = (float*)malloc(sizeof(float) * MATRIX_ROW_A * MATRIX_COL_A);
	bMatrixHost = (float*)malloc(sizeof(float) * MATRIX_ROW_B * MATRIX_COL_B);
	cMatrixHost = (float*)malloc(sizeof(float) * MATRIX_ROW_C * MATRIX_COL_C);
	resMatHost = (float*)malloc(sizeof(float) * MATRIX_ROW_C * MATRIX_COL_C);

	RandomInit(aMatrixHost, MATRIX_COL_A * MATRIX_ROW_A);
	RandomInit(bMatrixHost, MATRIX_COL_B * MATRIX_ROW_B);
	InitMatrixC(cMatrixHost, MATRIX_COL_C * MATRIX_ROW_C);

	MatrixMulHost(aMatrixHost, bMatrixHost, cMatrixHost, MATRIX_ROW_A, MATRIX_COL_A, MATRIX_COL_B);

	float *aMatDev, *bMatDev, *cMatDev;
	checkCudaErrors(hipMalloc(&aMatDev, sizeof(float) * MATRIX_ROW_A * MATRIX_COL_A));
	checkCudaErrors(hipMalloc(&bMatDev, sizeof(float) * MATRIX_ROW_B * MATRIX_COL_B));
	checkCudaErrors(hipMalloc(&cMatDev, sizeof(float) * MATRIX_ROW_C * MATRIX_COL_C));

	checkCudaErrors(hipMemcpy(aMatDev, aMatrixHost, sizeof(float) * MATRIX_ROW_A * MATRIX_COL_A, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(bMatDev, bMatrixHost, sizeof(float) * MATRIX_ROW_B * MATRIX_COL_B, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemset(cMatDev, 0, sizeof(float) * MATRIX_ROW_C * MATRIX_COL_C));
	
	const float alpha = 1.0f;
    const float beta  = 0.0f;
    hipblasHandle_t handle;
	
	checkCudaErrors(hipblasCreate(&handle));
	//col order swap a and b
	checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, MATRIX_COL_B, MATRIX_ROW_A, MATRIX_COL_A, &alpha, bMatDev, MATRIX_COL_B, aMatDev, MATRIX_COL_A, &beta, cMatDev, MATRIX_COL_C));
	checkCudaErrors(hipMemcpy(resMatHost, cMatDev, sizeof(float) * MATRIX_ROW_C * MATRIX_COL_C, hipMemcpyDeviceToHost));
	checkCudaErrors(hipblasDestroy(handle));

	float maxDiff = CheckResults(cMatrixHost, resMatHost, MATRIX_ROW_C * MATRIX_COL_C);

	printf("Max difference between DEVICE and HOST: %f\n", maxDiff);

	// for (int i = 0; i < MATRIX_COL_C * MATRIX_ROW_C; i++)
	// {
	// 	printf("res[%d]: %f / %f \n", i, resMatHost[i], cMatrixHost[i]);
	// }

	checkCudaErrors(hipFree(aMatDev));
	checkCudaErrors(hipFree(bMatDev));
	checkCudaErrors(hipFree(cMatDev));

	free(aMatrixHost);
	free(bMatrixHost);
	free(cMatrixHost);
	free(resMatHost);

	return 0;
}